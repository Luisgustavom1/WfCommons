#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cstdlib>  // For std::atoi
#include "gpu_benchmark.h"

// Kernel function to perform a simple workload
__global__ void simpleKernel(int* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * data[idx];  // Simple workload: squaring each element
    }
}

// Function to run the GPU benchmark
void runBenchmark(int max_work, int runtime_in_seconds) {
    int* h_data = new int[max_work];
    int* d_data;

    // Initialize data
    for (int i = 0; i < max_work; i++) {
        h_data[i] = i;
    }

    // Allocate GPU memory
    hipMalloc(&d_data, max_work * sizeof(int));

    // Copy data to GPU
    hipMemcpy(d_data, h_data, max_work * sizeof(int), hipMemcpyHostToDevice);

    // Start the timer
    auto start = std::chrono::high_resolution_clock::now();

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (max_work + threadsPerBlock - 1) / threadsPerBlock;

    // Run the workload loop until the specified runtime is reached
    while (std::chrono::duration_cast<std::chrono::seconds>(std::chrono::high_resolution_clock::now() - start).count() < runtime_in_seconds) {
        simpleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, max_work);
        hipDeviceSynchronize();  // Ensure the kernel has finished executing
    }

    // Copy results back to host (optional, just for validation)
    hipMemcpy(h_data, d_data, max_work * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_data);
    delete[] h_data;

    std::cout << "Benchmark completed!" << std::endl;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <max_work> <runtime_in_seconds>" << std::endl;
        return 1;
    }

    int max_work = std::atoi(argv[1]);
    int runtime_in_seconds = std::atoi(argv[2]);

    if (max_work <= 0 || runtime_in_seconds <= 0) {
        std::cerr << "Both max_work and runtime_in_seconds must be positive integers." << std::endl;
        return 1;
    }

    runBenchmark(max_work, runtime_in_seconds);

    return 0;
}
